/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void check_handshaking_gpu(int * strongNeighbor, int * matches, int numNodes) {
	/** YOUR CODE GOES BELOW **/
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	while( tid < numNodes )
	{
		if( tid == strongNeighbor[strongNeighbor[tid]] )
		{
			if( matches[tid] == -1 && matches[strongNeighbor[tid]] == -1 )
			{
				matches[tid] = strongNeighbor[tid];
				matches[strongNeighbor[tid]] = strongNeighbor[strongNeighbor[tid]];
			}
		}
		tid += ( blockDim.x * gridDim.x );
	}
	
	/** YOUR CODE GOES ABOVE **/
}
