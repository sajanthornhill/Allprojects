/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void exclusive_prefix_sum_gpu(int * oldSum, int * newSum, int distance, int numElements) {
	/** YOUR CODE GOES BELOW **/
int numThreads = blockDim.x * gridDim.x; 
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  
	int i;

	for(i = tid; i <= numElements; i += numThreads)
	{
		if(distance == 0 ){
			if(tid == 0){
				newSum[tid] = 0;	
			}
			else{
				
				newSum[tid] = oldSum[tid-1];
			}
		}
		else{ 
			if(tid - distance >= 0){ 
				newSum[tid] = oldSum[tid] + oldSum[tid-distance]; 
			

			}else{ 
				newSum[tid] = oldSum[tid];	
				
			}
		}

	}
	/** YOUR CODE GOES ABOVE **/
}
