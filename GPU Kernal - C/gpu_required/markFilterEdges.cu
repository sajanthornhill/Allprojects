/*
 **********************************************
 *  CS314 Principles of Programming Languages *
 *  Fall 2020                                 *
 **********************************************
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void markFilterEdges_gpu(int * src, int * dst, int * matches, int * keepEdges, int numEdges) {
	/** YOUR CODE GOES BELOW **/
	int numThreads = blockDim.x * gridDim.x; 
	int tid = blockDim.x * blockIdx.x + threadIdx.x;  
	int i;
	
	for(i = tid; i < numEdges; i += numThreads)
	{
	
	if(matches[src[tid]] == -1){
		if(matches[dst[tid]] == -1){
			
			keepEdges[tid] = 1;

		}else{
			keepEdges[tid] = 0;

		}
	}else{
		keepEdges[tid] = 0;
	} 


	}
	
	/** YOUR CODE GOES ABOVE **/
}
